#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

extern "C" {
#include "../vector.h"
#include "../triangle.h"  
#include "../tetraeder.h"
  
}
using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

__global__ void tet_acute_kernel(ptriangle triang, int dim, unsigned char * result, size_t N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>= N)
    return;
  arr3 P[5]; 
  arr3 cur_pt;
  arr3 normals[4];
  int t; //Temp
  cur_pt[0] = idx / ((dim+1) * (dim + 1));
  t = idx % ((dim+1) * (dim+1));
  cur_pt[1] = t / (dim + 1);
  cur_pt[2] = t % (dim + 1);
  
  subArr3(triang->vertices[1], triang->vertices[0], P[0]);
  subArr3(triang->vertices[2], triang->vertices[0], P[1]);
  subArr3(cur_pt, triang->vertices[0], P[2]);
  subArr3(triang->vertices[2], triang->vertices[1], P[3]);
  subArr3(cur_pt, triang->vertices[1], P[4]); 
  
  crossArr3(P[2],P[0], normals[2]); //Normal on facet 0,1,3
  crossArr3(P[0],P[1], normals[3]); //Normal on facet 0,1,2
  crossArr3(P[1],P[2], normals[1]); //Normal on facet 0,2,3
  crossArr3(P[4],P[3], normals[0]); //Normal on facet 1,2,3
  result[idx] = (dotArr3(normals[1], normals[2]) < 0 &
                 dotArr3(normals[2], normals[3]) < 0 &
                 dotArr3(normals[1], normals[3]) < 0 &
                 dotArr3(normals[0], normals[1]) < 0 &
                 dotArr3(normals[0], normals[2]) < 0 &
                 dotArr3(normals[0], normals[3]) < 0);
  
}



triangle rand_triangle(int dim) {
  triangle result;
  result.vertices[0][0] = rand() % dim;
  result.vertices[0][1] = rand() % dim;
  result.vertices[0][2] = rand() % dim;
  result.vertices[1][0] = rand() % dim;
  result.vertices[1][1] = rand() % dim;
  result.vertices[1][2] = rand() % dim;
  result.vertices[2][0] = rand() % dim;
  result.vertices[2][1] = rand() % dim;
  result.vertices[2][2] = rand() % dim;
  return result;
}

#define DIM 150
#define THREADS_BLOCK 512
int main(void)
{
  clock_t begin, end;
  hipEvent_t start, stop;
  float time_ms;
  triangle triang = rand_triangle(DIM);
  triangle * ptriang;
  arr3 dim = {DIM,DIM,DIM};
  cube_points cube_pts = gen_cube_points(dim);
  arr3 * cube_d;
  unsigned char * res_h, *res_d;
  timeval t1,t2;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  gettimeofday(&t1,NULL);
  begin = clock();
  checkCudaCall(hipMalloc(&res_d, cube_pts.len * sizeof(unsigned char)));
  checkCudaCall(hipMalloc(&ptriang, sizeof(triangle))); 
  res_h = (unsigned char *) malloc(cube_pts.len * sizeof(unsigned char));
  checkCudaCall(hipMemcpy(ptriang, &triang, sizeof(triangle), hipMemcpyHostToDevice));
  hipEventRecord(start,0);
  tet_acute_kernel <<< cube_pts.len/THREADS_BLOCK + 1  , THREADS_BLOCK >>> (ptriang, DIM, res_d, cube_pts.len);
  hipEventRecord(stop,0);  
  checkCudaCall(hipGetLastError());
  checkCudaCall(hipMemcpy(res_h, res_d, cube_pts.len * sizeof(unsigned char), hipMemcpyDeviceToHost));
  checkCudaCall(hipFree(res_d));
  end = clock();
  hipEventElapsedTime(&time_ms,start,stop);
  gettimeofday(&t2,NULL);
  printf("Wall time : %ld\n", ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))); 
  printf("Time taken on CPU: %f sec\n", float( (end - begin) )/ CLOCKS_PER_SEC);
  printf("Time taken op the GPU: %f msec\n", time_ms);
  gettimeofday(&t1,NULL);  
  begin = clock();
  for (int i=0; i<cube_pts.len; i++) {
    //printf("Cuda: %d %d\n", i, res_h[i]);
    tetra test_tetra;
    memcpy(test_tetra.vertices + 3, cube_pts.points + i, sizeof(arr3));
    memcpy(test_tetra.vertices , triang.vertices, 3 * sizeof(arr3)); 
    unsigned char  acute = (unsigned char) tetra_acute(&test_tetra);
    if (res_h[i] != acute) {
      printf("FAIL %d %d %d\n",i, res_h[i], acute );
    }
  }
  end  = clock();
  gettimeofday(&t2,NULL);
  printf("Time taken on the CPU: %f sec\n", float((end - begin)) / CLOCKS_PER_SEC);
  printf("Wall time : %ld\n", ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))); 
  free(cube_pts.points);
  free(res_h);
}
