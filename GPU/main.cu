#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
extern "C" {
#include "../vector.h"
#include "../triangle.h"  
#include "../tetraeder.h"
  
}
using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

__global__ void tet_acute_kernel(triangle triang, arr3 * new_vec, int * result, size_t N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>= N)
    return;
  
  arr3 P[5]; 
  arr3 normals[4]; 
  subArr3(triang.vertices[1], triang.vertices[0], P[0]);
  subArr3(triang.vertices[2], triang.vertices[0], P[1]);
  subArr3(triang.vertices[2], triang.vertices[1], P[3]);
  subArr3(new_vec[idx], triang.vertices[1], P[4]); 
  subArr3(new_vec[idx], triang.vertices[0], P[2]);
  
  crossArr3(P[2],P[0], normals[2]); //Normal on facet 0,1,3
  crossArr3(P[0],P[1], normals[3]); //Normal on facet 0,1,2
  crossArr3(P[1],P[2], normals[1]); //Normal on facet 0,2,3
  crossArr3(P[4],P[3], normals[0]); //Normal on facet 1,2,3

  result[idx] = (dotArr3(normals[1], normals[2]) < 0 &&
                 dotArr3(normals[1], normals[3]) < 0 &&
                 dotArr3(normals[0], normals[1]) < 0 &&
                 dotArr3(normals[0], normals[2]) < 0 &&
                 dotArr3(normals[0], normals[3]) < 0);
  
}



triangle rand_triangle(int dim) {
  triangle result;
  result.vertices[0][0] = rand() % dim;
  result.vertices[0][1] = rand() % dim;
  result.vertices[0][2] = rand() % dim;
  result.vertices[1][0] = rand() % dim;
  result.vertices[1][1] = rand() % dim;
  result.vertices[1][2] = rand() % dim;
  result.vertices[2][0] = rand() % dim;
  result.vertices[2][1] = rand() % dim;
  result.vertices[2][2] = rand() % dim;
  return result;
}

#define DIM 7

int main(void)
{
  triangle triang = rand_triangle(DIM);
  arr3 dim = {DIM,DIM,DIM};
  cube_points cube_pts = gen_cube_points(dim);
  
  arr3 * cube_d;
  int * res_h, *res_d;
  
  checkCudaCall(hipMalloc(&cube_d, cube_pts.len * sizeof(arr3)));
  checkCudaCall(hipMalloc(&res_d, cube_pts.len * sizeof(int)));
  
  res_h = (int *) malloc(cube_pts.len * sizeof(int));
  for (int i = 0; i < 5; i++)
    printf("%d %d\n", i, res_h[i]);
  
  checkCudaCall(hipMemcpy(cube_d, &cube_pts.points[0], cube_pts.len * sizeof(arr3), hipMemcpyHostToDevice));
  
  tet_acute_kernel <<< 1, cube_pts.len >>> (triang, cube_d, res_d, cube_pts.len);
  
  checkCudaCall(hipGetLastError());
  checkCudaCall(hipMemcpy(res_h, res_d, cube_pts.len * sizeof(int), hipMemcpyDeviceToHost));

  for (int i=0; i<5; i++) {
    printf("Cuda: %d %d\n", i, res_h[i]);
    tetra test_tetra;
    memcpy(test_tetra.vertices, &cube_pts.points[i], sizeof(arr3));
    memcpy(test_tetra.vertices + 1, triang.vertices, 3 * sizeof(arr3)); 
    printf("Normal: %d %d\n", i , tetra_acute(&test_tetra));
  }
   
  hipFree(cube_d);
  hipFree(res_d);
  free(cube_pts.points);
  free(res_h);
}
