#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

extern "C" {
#include "../vector.h"
#include "../triangle.h"  
#include "../tetraeder.h"
  
}
using namespace std;

#define DIM 150
#define THREADS_BLOCK 512



/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

__global__ void tet_acute_kernel(ptriangle triang, int dim, unsigned char * result, size_t N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx>= N)
    return;
  arr3 P[5]; 
  arr3 cur_pt;
  arr3 normals[4];
  int t; //Temp
  /*
   * Might as well copy the points once
   */
  cur_pt[0] = idx / ((dim+1) * (dim + 1));
  t = idx % ((dim+1) * (dim+1));
  cur_pt[1] = t / (dim + 1);
  cur_pt[2] = t % (dim + 1);
  
  subArr3(triang->vertices[1], triang->vertices[0], P[0]);
  subArr3(triang->vertices[2], triang->vertices[0], P[1]);
  subArr3(cur_pt, triang->vertices[0], P[2]);
  subArr3(triang->vertices[2], triang->vertices[1], P[3]);
  subArr3(cur_pt, triang->vertices[1], P[4]); 

  crossArr3(P[4],P[3], normals[0]); //Normal on facet 1,2,3  
  crossArr3(P[1],P[2], normals[1]); //Normal on facet 0,2,3
  crossArr3(P[2],P[0], normals[2]); //Normal on facet 0,1,3
  crossArr3(P[0],P[1], normals[3]); //Normal on facet 0,1,2
  //Normals[3] is the normal on the triangle plane
  t = dotArr3(normals[3], triang->vertices[0]); //Find the constant specific for this plane  
  
  result[idx] = ((dotArr3(normals[1], normals[2]) < 0) &
                 (dotArr3(normals[2], normals[3]) < 0) &
                 (dotArr3(normals[1], normals[3]) < 0) &
                 (dotArr3(normals[0], normals[1]) < 0) &
                 (dotArr3(normals[0], normals[2]) < 0) &
                 (dotArr3(normals[0], normals[3]) < 0)) //First bit
                 | 
                ((dotArr3(cur_pt, normals[3]) <  t) << 1); //second bit
                 

}


int facet_cube_acute_gpu(ptriangle triang, facet_acute_data * data, int mode, unsigned char * res_h) {
  /*
   * Every facet of an acute tetrahedron needs to be acute. If this facet is not even acute
   * we may directly stop checking this facet as it's never going to be part of an acute tetrahedron
   */
  if (!mat3_triangle_acute(triang->vertices)) 
    return 0;
    
    
  data->boundary_triangle = data->boundary_func(triang, data->cube->dim); //Boundary plane only needs acute tetra on 1 side
  int dim = data->cube->dim[0];
  data->acute_above = 0;
  data->acute_below = 0;
  data->tetra_above_len = 0;
  data->tetra_above = NULL;
  data->tetra_below_len = 0;
  data->tetra_below = NULL;
  
  
  
  for (size_t i = 0; i < data->cube->len; i++){
    if (res_h[i] == 1 && !data->acute_above ||
        res_h[i] == 3 && !data->acute_below) {
      arr3 cur_pt;
      int t;
      cur_pt[0] = i / ((dim+1) * (dim + 1));
      t = i % ((dim+1) * (dim+1));
      cur_pt[1] = t / (dim + 1);
      cur_pt[2] = t % (dim + 1);
      //All the facets must be in the acute_list
      if ((mode == FACET_ACUTE_LIST) && !facet_tetra_list(triang, cur_pt, data->acute_list))
        continue;   
      
      //Explicitly create a list of the acute tetrahedron
      if (mode == FACET_ACUTE_TETRA) {
        tetra test_tetra;
        memcpy(test_tetra.vertices, cur_pt, sizeof(arr3));
        memcpy(test_tetra.vertices + 1, triang->vertices, 3 * sizeof(arr3)); 
      
        if (res_h[i] == 1)
          tetra_add_array(test_tetra, &data->tetra_above, &data->tetra_above_len);
        else
          tetra_add_array(test_tetra, &data->tetra_below, &data->tetra_below_len);
      }
      //We only need to know if tetrahedron above and below acute
      else {
        if (res_h[i] == 1)
          data->acute_above = 1;
        else 
          data->acute_below = 1;
        if ((data->acute_above && data->acute_below) || data->boundary_triangle) {
          free(res_h);
          return 1;
        }
      }
    }
  }
  free(res_h);
  if (mode == FACET_ACUTE_TETRA) {
    data->acute_above = (data->tetra_above_len > 0);
    data->acute_below = (data->tetra_below_len > 0);
    if ((data->acute_above && data->acute_below) || (data->boundary_triangle && (data->acute_above || data->acute_below)))
      return 1;
  }
  return 0;  
}

int * facets_cube_acute_gpu(ptriangle triang, size_t n, facet_acute_data * data) {
  size_t len = data->cube->len;
  int * acute = (int *)  malloc(sizeof(int) * n);
  unsigned char * res_h, *res_d;
  ptriangle ptriang_d; 
  
  checkCudaCall(hipMalloc(&res_d, len * sizeof(unsigned char)));
  checkCudaCall(hipMalloc(&ptriang_d, sizeof(triangle))); 
  res_h = (unsigned char *) malloc(len * sizeof(unsigned char));  
  
  for (size_t i = 0; i < n; i++) {
    checkCudaCall(hipMemcpy(ptriang_d, triang + i, sizeof(triangle), hipMemcpyHostToDevice));
    tet_acute_kernel <<< len/THREADS_BLOCK + 1  , THREADS_BLOCK >>> (ptriang_d, data->cube->dim[0], res_d, len);
    checkCudaCall(hipMemcpy(res_h, res_d, len * sizeof(unsigned char), hipMemcpyDeviceToHost));
    acute[i] = facet_cube_acute_gpu(triang + i,data,FACET_ACUTE, res_h);
  }
  
  checkCudaCall(hipFree(res_d));
  checkCudaCall(hipFree(ptriang_d));
  return acute;

}
triangle rand_triangle(int dim) {
  triangle result;
  result.vertices[0][0] = rand() % dim;
  result.vertices[0][1] = rand() % dim;
  result.vertices[0][2] = rand() % dim;
  result.vertices[1][0] = rand() % dim;
  result.vertices[1][1] = rand() % dim;
  result.vertices[1][2] = rand() % dim;
  result.vertices[2][0] = rand() % dim;
  result.vertices[2][1] = rand() % dim;
  result.vertices[2][2] = rand() % dim;
  return result;
}
#define SIZE_LIST 100000
int main(void)
{
  clock_t begin, end;
  timeval t1,t2;
  triangle * triangles = (triangle * ) malloc(sizeof(triangle) * SIZE_LIST);
  int * acute;
  for (int i = 0; i < SIZE_LIST; i++)
    triangles[i] = rand_triangle(DIM);
  //triangle triang = rand_triangle(DIM);
  arr3 dim = {DIM,DIM,DIM};
  cube_points cube_pts = gen_cube_points(dim);
  facet_acute_data parameters;
  parameters.cube = &cube_pts;
  parameters.boundary_func = &triangle_boundary_cube;
  printf("Triangle: \n");
  gettimeofday(&t1,NULL);
  begin = clock();
  acute = facets_cube_acute_gpu(triangles,SIZE_LIST,&parameters);
  end = clock();
  printf("Acute_GPU: %d\n", acute);
  gettimeofday(&t2,NULL);
  printf("Wall time : %ld\n", ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000); 
  printf("Time taken on CPU: %f sec\n", float( (end - begin) )/ CLOCKS_PER_SEC);
  gettimeofday(&t1,NULL);
  begin = clock();
  for (int i =0; i < SIZE_LIST; i++)
    acute[i] = facet_cube_acute(triangles + i,&parameters,FACET_ACUTE);
  end = clock();
  printf("Acute: %d\n", acute);
  gettimeofday(&t2,NULL);
  printf("Wall time : %ld\n", ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000); 
  printf("Time taken on CPU: %f sec\n", float( (end - begin) )/ CLOCKS_PER_SEC);
  free(cube_pts.points);

}
